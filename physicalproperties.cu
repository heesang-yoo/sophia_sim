// === Global Variable Definitions ===
// These variables are declared as 'extern' in SophiaSim.cuh.
// Here we provide the actual definitions and memory allocation.
// Only one definition is allowed across the entire program.

#include "physicalproperties.cuh"
#include <hip/hip_runtime.h>

Real host_Tab_T[table_size];
Real host_Tab_h[table_size];
Real host_Tab_k[table_size];
Real host_Tab_cp[table_size];
Real host_Tab_vis[table_size];

__constant__ Real k_Tab_T[table_size];
__constant__ Real k_Tab_h[table_size];
__constant__ Real k_Tab_k[table_size];
__constant__ Real k_Tab_cp[table_size];
__constant__ Real k_Tab_vis[table_size];

__constant__ int k_table_index[10];
__constant__ int k_table_size[10];

void initializePropertyTables()
{
    hipMemcpyToSymbol(HIP_SYMBOL(k_Tab_T), host_Tab_T, sizeof(Real) * table_size);
    hipMemcpyToSymbol(HIP_SYMBOL(k_Tab_h), host_Tab_h, sizeof(Real) * table_size);
    hipMemcpyToSymbol(HIP_SYMBOL(k_Tab_k), host_Tab_k, sizeof(Real) * table_size);
    hipMemcpyToSymbol(HIP_SYMBOL(k_Tab_cp), host_Tab_cp, sizeof(Real) * table_size);
    hipMemcpyToSymbol(HIP_SYMBOL(k_Tab_vis), host_Tab_vis, sizeof(Real) * table_size);

    hipMemcpyToSymbol(HIP_SYMBOL(k_table_index), host_table_index, sizeof(int) * 10);
    hipMemcpyToSymbol(HIP_SYMBOL(k_table_size), host_table_size, sizeof(int) * 10);
}
