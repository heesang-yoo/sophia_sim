#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string>
#include <algorithm>
#include <math.h>
#include <time.h>
#include <iostream>
#include <dirent.h>
#include <sys/stat.h>
#include <ctime>
#include <hipcub/hipcub.hpp>

#include "SophiaSim.cuh"

// === Global Variable Definitions ===
// These variables are declared as 'extern' in SophiaSim.cuh.
// Here we provide the actual definitions and memory allocation.
// Only one definition is allowed across the entire program.

int_t vii[vii_size];
Real vif[vif_size];

__constant__ int_t k_vii[vii_size];
__constant__ Real k_vif[vif_size];

part1* HP1;
part1* DHP1[Max_GPU];

Real space;
int Nsx = 0;
int Nsz = 0;
int buffer_size = 0;

int num_plot_data;
char plot_data[20][20];

#include "physicalproperties.cuh"
#include "function_init.cuh"
#include "functions_NNPS.cuh"
#include "functions_PROP.cuh"
#include "functions_KNL.cuh"
#include "functions_PREP.cuh"
#include "functions_MASS.cuh"
#include "functions_INTERACTION.cuh"
#include "functions_TIME.cuh"
#include "functions_TIME_ISPH.cuh"
#include "functions_OUTPUT.cuh"
#include "functions_PST.cuh"
#include "functions_ALE.cuh"
#include "functions_BC.cuh"
#include "functions_PPE.cuh"
#include "ISPH_Calc.cuh"

void SophiaSim::initialize(int argc, char** argv) {
    memset(vii, 0, sizeof(int_t) * vii_size);
    memset(vif, 0, sizeof(Real) * vif_size);

    if (argc < 2) {
        printf("Usage: %s <number_of_GPUs>\n", argv[0]);
        exit(-1);
    }
    ngpu = atoi(argv[1]);

    read_solv_input(vii, vif, "./input/solv.txt");
    read_table("./input/data.txt");
}

void SophiaSim::loadInput() {
    char INPUT_FILE_NAME[128] = "./input/input.txt";
    num_part = gpu_count_particle_numbers2(INPUT_FILE_NAME);

    HP1 = (part1*)malloc(num_part * sizeof(part1));
    memset(HP1, 0, sizeof(part1) * num_part);
    read_input(HP1);
}

void SophiaSim::setupDomain() {
    find_minmax(vii, vif, HP1);
    search_kappa = kappa;
    Real h0 = h_max;
    Real cell_reduction_factor = 1.1;
    search_incr_factor = 1.0;

    dcell = (cell_reduction_factor * kappa * h0) / static_cast<Real>(ncell_init);
    if (dcell <= 0.0) {
        printf("\u274c dcell must be positive. Current: %.12f\n", dcell);
        exit(EXIT_FAILURE);
    }

    NI = static_cast<int>((x_max - x_min) / dcell + 1.0);
    NJ = static_cast<int>((y_max - y_min) / dcell + 1.0);
    NK = static_cast<int>((z_max - z_min) / dcell + 1.0);

    int tNx = (int)((x_max - x_min) / (h0 / h_coeff)) + 1;
    int tNz = (int)((z_max - z_min) / (h0 / h_coeff)) + 1;

    calc_area = ceil((float)NI / ngpu);

    if (ngpu > 1) {
        num_p2p = (int)(num_part * 4 / NI * C_p2p);
        num_part2 = (int)((num_part / ngpu) * 1.2) + 2 * num_p2p;
    } else {
        if (open_boundary > 0) {
            Real space = h0 / h_coeff * 0.5;
            Nsx = (int)((x_max - x_min) / space) + 1;
            Nsz = (int)((y_max - y_min) / space) + 1;
            buffer_size = (Nsx * Nsz);
        }

        buffer_size = 0;
        num_part2 = num_part + buffer_size;
        num_part3 = num_part2;

        if (aps_solv) {
            if (dim == 2) {
                num_part2 = num_part + num_part * 4;
                num_part3 = num_part2 + buffer_size;
            }
            if (dim == 3)
                num_part3 = 4 * num_part + buffer_size;
        }

        if (open_boundary > 0) {
            printf("Buffer memory size: %d\n", buffer_size);
            printf("Inlet particle generation frequency: %d steps\n",
                   (int)((h0 / h_coeff) / (Inlet_Velocity * dt)));
        }
    }
}

void SophiaSim::runSimulation() {
    int tid = 0;
    printf("[DEBUG] tid = %d\n", tid);
    ISPH_Calc((void*)&tid);
}

void SophiaSim::cleanup() {
    if (HP1) free(HP1);
}