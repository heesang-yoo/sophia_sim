#include "SophiaSim.cuh"

int main(int argc, char** argv) {
    SophiaSim sim;
    sim.initialize(argc, argv);
    sim.loadInput();
    sim.setupDomain();
    sim.runSimulation();
    sim.cleanup();
    return 0;
}