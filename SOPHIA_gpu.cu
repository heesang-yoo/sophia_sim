#include <stdio.h>
#include <string>
#include <algorithm>
#include <math.h>
#include <time.h>
#include <hipcub/hipcub.hpp>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "hip/hip_runtime.h"
#include ""

#include "Variable_Type.cuh"
#include "Parameters.cuh"
#include "class_Cuda_Particle_Array.cuh"

//------------------------------------------------------------------
// Global Variable Declarations
//------------------------------------------------------------------

// Solver parameters (see function_init.cuh)
// Host
int_t vii[vii_size];
Real vif[vif_size];

// Device (solver options)
__constant__ int_t k_vii[vii_size];
__constant__ Real k_vif[vif_size];

// Device (property tables)
__constant__ Real k_Tab_T[table_size];    // Temperature table
__constant__ Real k_Tab_h[table_size];    // Enthalpy table
__constant__ Real k_Tab_k[table_size];    // Conductivity table
__constant__ Real k_Tab_cp[table_size];   // Specific heat table
__constant__ Real k_Tab_vis[table_size];  // Viscosity table
__constant__ int k_table_index[10];       // Start index for each table
__constant__ int k_table_size[10];        // Number of entries for each table

// Particle arrays
part1* HP1;                       // All host particles
part1* DHP1[Max_GPU];             // Host-side arrays per GPU (for partitioning)

// Host-side tables
Real host_Tab_T[table_size];
Real host_Tab_h[table_size];
Real host_Tab_k[table_size];
Real host_Tab_cp[table_size];
Real host_Tab_vis[table_size];

int host_table_index[10];
int host_table_size[10];

// Open boundary (buffer zone for inflow/outflow)
Real space;           // Virtual grid spacing for open boundary inlet
int Nsx = 0;          // Number of grid points along x for open boundary
int Nsz = 0;          // Number of grid points along z (or y) for open boundary
int buffer_size = 0;  // Additional memory for generated buffer particles

__device__ int num_buffer[1];     // Total number of buffer particles (needed for Open Boundary/APS models)

// Plot Data
int num_plot_data;                // Number of plot variables
char plot_data[20][20];           // Names of variables to plot

// Function header includes (simulation core logic)
#include "function_init.cuh"
#include "functions_NNPS.cuh"
#include "functions_PROP.cuh"
#include "functions_KNL.cuh"
#include "functions_PREP.cuh"
#include "functions_TIME_ISPH.cuh"
#include "functions_ALE.cuh"
#include "functions_BC.cuh"
#include "functions_PPE.cuh"
#include "functions_OUTPUT.cuh"
#include "functions_PST.cuh"
#include "ISPH_Calc.cuh"

int main(int argc, char **argv)
{
    // Initialize solver parameters
    memset(vii, 0, sizeof(int_t) * vii_size);
    memset(vif, 0, sizeof(Real) * vif_size);

    ngpu = atoi(argv[1]); // Number of GPUs (from command-line)

    char fn[64], fn2[64];
    strcpy(fn, "./input/solv.txt");
    strcpy(fn2, "./input/data.txt");

    read_solv_input(vii, vif, fn);    // Read solver option input file
    read_table(fn2);                  // Read property table

    // Print GPU device properties
    {
        int_t gcount, i;
        struct hipDeviceProp_t prop;
        hipGetDeviceCount(&gcount);

        for (i = 0; i < gcount; i++) {
            hipGetDeviceProperties(&prop, i);
            printf("### GPU DEVICE PROPERTIES.................................\n\n");
            printf("    Name: %s\n", prop.name);
            printf("    Compute capability: %d.%d\n", prop.major, prop.minor);
            printf("    Clock rate: %d\n", prop.clockRate);
            printf("    Total global memory: %ld\n", prop.totalGlobalMem);
            printf("    Total constant memory: %d\n", prop.totalConstMem);
            printf("    Multiprocessor count: %d\n", prop.multiProcessorCount);
            printf("    Shared mem per block: %d\n", prop.sharedMemPerBlock);
            printf("    Registers per block: %d\n", prop.regsPerBlock);
            printf("    Threads in warp: %d\n", prop.warpSize);
            printf("    Max threads per block: %d\n", prop.maxThreadsPerBlock);
            printf("    Max thread dimensions: %d,%d,%d\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
            printf("    Max grid dimensions: %d,%d,%d\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
            printf("...........................................................\n\n");
        }
    }

    printf(" ------------------------------------------------------------\n");
    printf(" SOPHIA_gpu v.1.0 \n");
    printf(" Developed by E.S. Kim, Y.B. Jo, S.H. Park\n");
    printf(" 2017. 02. 20 \n");
    printf(" Optimized by Y.W. Sim, CoCoLink Inc.\n");
    printf(" 2018, 2019(C) \n");
    printf(" Restructured & Innovated by Eung Soo Kim, Hee Sang Yoo, Young Beom Jo, Hae Yoon Choi, Su-San Park, Jin Woo Kim, Yelyn Ahn, Tae Soo Choi\n");
    printf(" ESLAB, SEOUL NATIONAL UNIVERSITY, SOUTH KOREA.\n");
    printf(" 2019. 08. 08 \n");
    printf("------------------------------------------------------------\n\n");

    // Count the number of particles in the domain
    char INPUT_FILE_NAME[128];
    strcpy(INPUT_FILE_NAME, "./input/input.txt");
    num_part = gpu_count_particle_numbers2(INPUT_FILE_NAME);

    // Allocate and initialize host memory for particles
    HP1 = (part1 *)malloc(num_part * sizeof(part1));
    memset(HP1, 0, sizeof(part1) * num_part);

    // Read initial particle data from file
    read_input(HP1);

    // Domain and cell decomposition (spatial partitioning for neighbor search)
    Real cell_reduction_factor = 1.1;
    search_incr_factor = 1.0;
    search_kappa = kappa;
    find_minmax(vii, vif, HP1);

    Real h0 = h_max;
    dcell = cell_reduction_factor * kappa * h0 / ncell_init;

    NI = (int)((x_max - x_min) / dcell) + 1;
    NJ = (int)((y_max - y_min) / dcell) + 1;
    NK = (int)((z_max - z_min) / dcell) + 1;

    int tNx = (int)((x_max - x_min) / (h0 / h_coeff)) + 1;
    int tNz = (int)((z_max - z_min) / (h0 / h_coeff)) + 1;

    // Open boundary setup (for inflow/outflow simulation)
    if (open_boundary > 0) {
        space = h0 / h_coeff * 0.5;
        Nsx = (int)((x_max - x_min) / space) + 1;
        Nsz = (int)((z_max - z_min) / space) + 1;
        buffer_size = Nsx * Nsz;

        printf("\n\n----------------------------------------------------\n");
        printf("Open Boundary Space = %f \n", space);
        printf("Buffer memory size: %d    ratio: %.2f \n\n", buffer_size, (Real)buffer_size / (tNx * tNz));
        printf("Frequency of inlet particle generation: %d steps\n", (int)((h0 / h_coeff) / (Inlet_Velocity * dt)));
        printf("----------------------------------------------------\n");
    } else {
        buffer_size = 0;
    }

    // Particle counts with/without buffer zone and APS extension
    num_part2 = num_part + buffer_size;
    num_part3 = num_part2;

    if (aps_solv) {
        if (dim == 2) {
            num_part2 = num_part + num_part * 4;
            num_part3 = num_part2 + buffer_size;
        }
        if (dim == 3)
            num_part3 = 4 * num_part + buffer_size;
    }

    // Start the main solver (single GPU)
    int tid = 0;
    ISPH_Calc((void *)&tid);

    // Free allocated host memory
    free(HP1);
    return 0;
}
