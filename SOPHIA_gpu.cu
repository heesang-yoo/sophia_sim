#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string>
#include <algorithm>
#include <math.h>
#include <time.h>
#include <hipcub/hipcub.hpp>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "hip/hip_runtime.h"
#include ""

#include "Variable_Type.cuh"
#include "Parameters.cuh"
#include "class_Cuda_Particle_Array.cuh"

//-------------------------------------------------------------------------------------------------
// ##. Global Variable Declarations
//-------------------------------------------------------------------------------------------------

// (Memory Allocation & Initialization)
// Host
void* malloc_and_zero(size_t size) {
    void* ptr = malloc(size);
    memset(ptr, 0, size);
    return ptr;
}
// Device
void cuda_malloc_and_zero(void** devPtr, size_t size) {
    hipMalloc(devPtr, size);
    hipMemset(*devPtr, 0, size);
}

// (solver options)
// Host
int_t *vii = (int_t*)malloc_and_zero(sizeof(int_t) * vii_size);
Real *vif = (Real*)malloc_and_zero(sizeof(Real) * vif_size);
// Device
__constant__ int_t k_vii[vii_size];
__constant__ Real k_vif[vif_size];

// (output options)
int num_plot_data;                // Number of plot variables
char plot_data[20][20];           // Names of variables to plot

// (table properties)
// Host
Real host_Tab_T[table_size];
Real host_Tab_h[table_size];
Real host_Tab_k[table_size];
Real host_Tab_cp[table_size];
Real host_Tab_vis[table_size];
int host_table_index[10];
int host_table_size[10];
// Device
__constant__ Real k_Tab_T[table_size];    // Temperature table
__constant__ Real k_Tab_h[table_size];    // Enthalpy table
__constant__ Real k_Tab_k[table_size];    // Conductivity table
__constant__ Real k_Tab_cp[table_size];   // Specific heat table
__constant__ Real k_Tab_vis[table_size];  // Viscosity table
__constant__ int k_table_index[10];       // Start index for each table
__constant__ int k_table_size[10];        // Number of entries for each table

// (OBC properties)
Real space;           // Virtual grid spacing for open boundary inlet
int Nsx = 0;          // Number of grid points along x for open boundary
int Nsz = 0;          // Number of grid points along z (or y) for open boundary
int buffer_size = 0;  // Additional memory for generated buffer particles

// (Particle Arrays)
part1* HP1;                       // All host particles
part1* DHP1[Max_GPU];             // Host-side arrays per GPU (for partitioning)

// Function header includes (simulation core logic)
#include "function_init.cuh"
#include "functions_NNPS.cuh"
#include "functions_PROP.cuh"
#include "functions_KNL.cuh"
#include "functions_PREP.cuh"
#include "functions_TIME_ISPH.cuh"
#include "functions_ALE.cuh"
#include "functions_BC.cuh"
#include "functions_PPE.cuh"
#include "functions_OUTPUT.cuh"
#include "functions_PST.cuh"

int main(int argc, char **argv)
{
    // Initialize solver parameters
    memset(vii, 0, sizeof(int_t) * vii_size);
    memset(vif, 0, sizeof(Real) * vif_size);

    ngpu = atoi(argv[1]); // Number of GPUs (from command-line)

    char fn[64], fn2[64];
    strcpy(fn, "./input/solv.txt");
    strcpy(fn2, "./input/data.txt");

    read_solv_input(vii, vif, fn);    // Read solver option input file
    read_table(fn2);                  // Read property table

    // Print GPU device properties
    {
        int_t gcount, i;
        struct hipDeviceProp_t prop;
        hipGetDeviceCount(&gcount);

        for (i = 0; i < gcount; i++) {
            hipGetDeviceProperties(&prop, i);
            printf("### GPU DEVICE PROPERTIES.................................\n\n");
            printf("    Name: %s\n", prop.name);
            printf("    Compute capability: %d.%d\n", prop.major, prop.minor);
            printf("    Clock rate: %d\n", prop.clockRate);
            printf("    Total global memory: %ld\n", prop.totalGlobalMem);
            printf("    Total constant memory: %d\n", prop.totalConstMem);
            printf("    Multiprocessor count: %d\n", prop.multiProcessorCount);
            printf("    Shared mem per block: %d\n", prop.sharedMemPerBlock);
            printf("    Registers per block: %d\n", prop.regsPerBlock);
            printf("    Threads in warp: %d\n", prop.warpSize);
            printf("    Max threads per block: %d\n", prop.maxThreadsPerBlock);
            printf("    Max thread dimensions: %d,%d,%d\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
            printf("    Max grid dimensions: %d,%d,%d\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
            printf("...........................................................\n\n");
        }
    }

    printf(" ------------------------------------------------------------\n");
    printf(" SOPHIA_gpu v.1.0 \n");
    printf(" Developed by E.S. Kim, Y.B. Jo, S.H. Park\n");
    printf(" 2017. 02. 20 \n");
    printf(" Optimized by Y.W. Sim, CoCoLink Inc.\n");
    printf(" 2018, 2019(C) \n");
    printf(" Restructured & Innovated by Eung Soo Kim, Hee Sang Yoo, Young Beom Jo, Hae Yoon Choi, Su-San Park, Jin Woo Kim, Yelyn Ahn, Tae Soo Choi\n");
    printf(" ESLAB, SEOUL NATIONAL UNIVERSITY, SOUTH KOREA.\n");
    printf(" 2019. 08. 08 \n");
    printf("------------------------------------------------------------\n\n");

    // Count the number of particles in the domain
    char INPUT_FILE_NAME[128];
    strcpy(INPUT_FILE_NAME, "./input/input.txt");
    num_part = gpu_count_particle_numbers2(INPUT_FILE_NAME);

    // Allocate and initialize host memory for particles
    HP1 = (part1 *)malloc(num_part * sizeof(part1));
    memset(HP1, 0, sizeof(part1) * num_part);

    // Read initial particle data from file
    read_input(HP1);

    // Domain and cell decomposition (spatial partitioning for neighbor search)
    Real cell_reduction_factor = 1.1;
    search_incr_factor = 1.0;
    search_kappa = kappa;
    find_minmax(vii, vif, HP1);

    Real h0 = h_max;
    dcell = cell_reduction_factor * kappa * h0 / ncell_init;

    NI = (int)((x_max - x_min) / dcell) + 1;
    NJ = (int)((y_max - y_min) / dcell) + 1;
    NK = (int)((z_max - z_min) / dcell) + 1;

    int tNx = (int)((x_max - x_min) / (h0 / h_coeff)) + 1;
    int tNz = (int)((z_max - z_min) / (h0 / h_coeff)) + 1;

    // Open boundary setup (for inflow/outflow simulation)
    if (open_boundary > 0) {
        space = h0 / h_coeff * 0.5;
        Nsx = (int)((x_max - x_min) / space) + 1;
        Nsz = (int)((z_max - z_min) / space) + 1;
        buffer_size = Nsx * Nsz;

        printf("\n\n----------------------------------------------------\n");
        printf("Open Boundary Space = %f \n", space);
        printf("Buffer memory size: %d    ratio: %.2f \n\n", buffer_size, (Real)buffer_size / (tNx * tNz));
        printf("Frequency of inlet particle generation: %d steps\n", (int)((h0 / h_coeff) / (Inlet_Velocity * dt)));
        printf("----------------------------------------------------\n");
    } else {
        buffer_size = 0;
    }

    // Particle counts with/without buffer zone and APS extension
    num_part2 = num_part + buffer_size;
    num_part3 = num_part2;

    if (aps_solv) {
        if (dim == 2) {
            num_part2 = num_part + num_part * 4;
            num_part3 = num_part2 + buffer_size;
        }
        if (dim == 3)
            num_part3 = 4 * num_part + buffer_size;
    }

    // Start the main solver (single GPU)
    ///////////////////////////////////////////////////////////////////////////////////////////////////////
    int tid = 0;
    // int tid = *(int*)arg;
    num_cells = clc_num_cells();
    count = floor(time/dt + 0.5);
    hipSetDevice(tid);

    // --- Host 메모리 선언 및 할당 ---
    part1* file_P1 = (part1*)malloc_and_zero(sizeof(part1) * num_part3);
    part2* file_P2 = (part2*)malloc_and_zero(sizeof(part2) * num_part3);
    part3* file_P3 = (part3*)malloc_and_zero(sizeof(part3) * num_part3);

    Real *max_umag0 = (Real*)malloc_and_zero(sizeof(Real));
    Real *max_rho0  = (Real*)malloc_and_zero(sizeof(Real));
    Real *max_ftotal0 = (Real*)malloc_and_zero(sizeof(Real));
    Real *max_phi0  = (Real*)malloc_and_zero(sizeof(Real));
    Real *dt10 = (Real*)malloc_and_zero(sizeof(Real));
    Real *dt20 = (Real*)malloc_and_zero(sizeof(Real));
    Real *dt30 = (Real*)malloc_and_zero(sizeof(Real));
    Real *dt40 = (Real*)malloc_and_zero(sizeof(Real));
    Real *dt50 = (Real*)malloc_and_zero(sizeof(Real));

    // --- Device 메모리 선언 및 할당 ---
    int_t *g_idx, *p_idx, *g_idx_in, *p_idx_in, *g_str, *g_end;
    cuda_malloc_and_zero((void**)&g_idx,     sizeof(int_t) * num_part3);
    cuda_malloc_and_zero((void**)&p_idx,     sizeof(int_t) * num_part3);
    cuda_malloc_and_zero((void**)&g_idx_in,  sizeof(int_t) * num_part3);
    cuda_malloc_and_zero((void**)&p_idx_in,  sizeof(int_t) * num_part3);
    cuda_malloc_and_zero((void**)&g_str,     sizeof(int_t) * num_cells);
    cuda_malloc_and_zero((void**)&g_end,     sizeof(int_t) * num_cells);

    part1 *dev_P1, *dev_SP1;
    part2 *dev_P2, *dev_SP2;
    part3 *dev_SP3;
    cuda_malloc_and_zero((void**)&dev_P1,   sizeof(part1) * num_part3);
    cuda_malloc_and_zero((void**)&dev_SP1,  sizeof(part1) * num_part3);
    cuda_malloc_and_zero((void**)&dev_P2,   sizeof(part2) * num_part3);
    cuda_malloc_and_zero((void**)&dev_SP2,  sizeof(part2) * num_part3);
    cuda_malloc_and_zero((void**)&dev_SP3,  sizeof(part3) * num_part3);

    Real *max_rho, *max_umag, *max_ft, *max_phi;
    cuda_malloc_and_zero((void**)&max_rho,    sizeof(Real)*num_part3);
    cuda_malloc_and_zero((void**)&max_umag,   sizeof(Real)*num_part3);
    cuda_malloc_and_zero((void**)&max_ft,     sizeof(Real)*num_part3);
    cuda_malloc_and_zero((void**)&max_phi,    sizeof(Real)*num_part3);

    Real *d_max_umag0, *d_max_rho0, *d_max_ftotal0, *d_max_phi0;
    cuda_malloc_and_zero((void**)&d_max_umag0,   sizeof(Real));
    cuda_malloc_and_zero((void**)&d_max_rho0,    sizeof(Real));
    cuda_malloc_and_zero((void**)&d_max_ftotal0, sizeof(Real));
    cuda_malloc_and_zero((void**)&d_max_phi0,    sizeof(Real));

    Real *dt1, *dt2, *dt3, *dt4, *dt5;
    cuda_malloc_and_zero((void**)&dt1, sizeof(Real)*num_part3);
    cuda_malloc_and_zero((void**)&dt2, sizeof(Real)*num_part3);
    cuda_malloc_and_zero((void**)&dt3, sizeof(Real)*num_part3);
    cuda_malloc_and_zero((void**)&dt4, sizeof(Real)*num_part3);
    cuda_malloc_and_zero((void**)&dt5, sizeof(Real)*num_part3);

    Real *d_dt10, *d_dt20, *d_dt30, *d_dt40, *d_dt50;
    cuda_malloc_and_zero((void**)&d_dt10, sizeof(Real));
    cuda_malloc_and_zero((void**)&d_dt20, sizeof(Real));
    cuda_malloc_and_zero((void**)&d_dt30, sizeof(Real));
    cuda_malloc_and_zero((void**)&d_dt40, sizeof(Real));
    cuda_malloc_and_zero((void**)&d_dt50, sizeof(Real));

    void* dev_sort_storage = nullptr;
    void* dev_max_storage = nullptr;
    size_t sort_storage_bytes = 0;
    size_t max_storage_bytes = 0;

	hipcub::DeviceRadixSort::SortPairs(dev_sort_storage,sort_storage_bytes,g_idx_in,g_idx,p_idx_in,p_idx,num_part3);
	hipcub::DeviceReduce::Max(dev_max_storage,max_storage_bytes,max_umag,d_max_umag0,num_part3);
	hipMalloc((void**)&dev_sort_storage,sort_storage_bytes);
	hipMalloc((void**)&dev_max_storage,max_storage_bytes);

    hipMemcpyToSymbol(HIP_SYMBOL(k_vii), vii, sizeof(int_t) * vii_size);
    hipMemcpyToSymbol(HIP_SYMBOL(k_vif), vif, sizeof(Real) * vif_size);
    hipMemcpyToSymbol(HIP_SYMBOL(k_Tab_T), host_Tab_T, sizeof(Real) * table_size);
    hipMemcpyToSymbol(HIP_SYMBOL(k_Tab_h), host_Tab_h, sizeof(Real) * table_size);
    hipMemcpyToSymbol(HIP_SYMBOL(k_Tab_k), host_Tab_k, sizeof(Real) * table_size);
    hipMemcpyToSymbol(HIP_SYMBOL(k_Tab_cp), host_Tab_cp, sizeof(Real) * table_size);
    hipMemcpyToSymbol(HIP_SYMBOL(k_Tab_vis), host_Tab_vis, sizeof(Real) * table_size);
    hipMemcpyToSymbol(HIP_SYMBOL(k_table_index), host_table_index, sizeof(int) * 10);
    hipMemcpyToSymbol(HIP_SYMBOL(k_table_size), host_table_size, sizeof(int) * 10);

    DHP1[tid] = (part1*)malloc(num_part3 * sizeof(part1));
    memset(DHP1[tid], 0, sizeof(part1) * num_part3);
    for (int i = 0; i < num_part3; i++) DHP1[tid][i].i_type = 3;
    c_initial_inner_outer_particle_single(HP1, DHP1[tid], tid);
    hipMemcpy(dev_P1, DHP1[tid], num_part3 * sizeof(part1), hipMemcpyHostToDevice);

	if (tid == 0) {
		printf("-----------------------------------------------------------\n");
		printf("SOPHIA SPH Simulation: Input Summary\n");
		printf("-----------------------------------------------------------\n");
		printf("Total Particles     : %d\n", num_part);
		printf("Device Particles    : %d\n", num_part3);
		printf("Grid (NI,NJ,NK)     : %d, %d, %d\n", NI, NJ, NK);
		printf("Domain: X [%f, %f]\n", x_min, x_max);
		printf("Domain: Y [%f, %f]\n", y_min, y_max);
		printf("Domain: Z [%f, %f]\n", z_min, z_max);
		printf("Cell Size (dcell)   : %f\n", dcell);
		printf("Cells/GPU (X-dir)   : %d\n", calc_area);
		printf("-----------------------------------------------------------\n");
		printf("Simulation Start!\n");
		printf("-----------------------------------------------------------\n\n");
	}

	//-------------------------------------------------------------------------------------------------
	// ##. Main Loop
	//-------------------------------------------------------------------------------------------------

    int_t N = 2 * time_end / time_output;
    Real *Cdp    = (Real*)malloc_and_zero(sizeof(Real) * N);
    Real *Cdv    = (Real*)malloc_and_zero(sizeof(Real) * N);
    Real *Cd     = (Real*)malloc_and_zero(sizeof(Real) * N);
    Real *Clp    = (Real*)malloc_and_zero(sizeof(Real) * N);
    Real *Clv    = (Real*)malloc_and_zero(sizeof(Real) * N);
    Real *Cl     = (Real*)malloc_and_zero(sizeof(Real) * N);
    Real *P0     = (Real*)malloc_and_zero(sizeof(Real) * N);
    Real *P1     = (Real*)malloc_and_zero(sizeof(Real) * N);
    Real *P2     = (Real*)malloc_and_zero(sizeof(Real) * N);
    Real *P3     = (Real*)malloc_and_zero(sizeof(Real) * N);
    Real *time0  = (Real*)malloc_and_zero(sizeof(Real) * N);
    int_t *plotcount = (int_t*)malloc_and_zero(sizeof(int_t) * 1);

    clock_t start=clock();

    while(1){
    
        //-------------------------------------------------------------------------------------------------
        // Time-step Control
        //-------------------------------------------------------------------------------------------------
        if(tid==0){
                dim3 b,t;
                t.x=128;
                b.x=(num_part3-1)/t.x+1;
                int_t s=sizeof(int_t)*(t.x+1);

                kernel_copy_max_timestep<<<b,t>>>(dev_P1,dev_SP2,dev_SP3,dt1, dt2, dt3, dt4, dt5);
                hipDeviceSynchronize();
                // Find Max Velocity & Force using CUB - TID=0
                hipcub::DeviceReduce::Max(dev_max_storage,max_storage_bytes,max_phi,d_max_phi0,num_part3);
                hipcub::DeviceReduce::Max(dev_max_storage,max_storage_bytes,max_ft,d_max_ftotal0,num_part3);
                hipcub::DeviceReduce::Max(dev_max_storage,max_storage_bytes,max_umag,d_max_umag0,num_part3);
                hipDeviceSynchronize();
                hipMemcpy(max_phi0,d_max_phi0,sizeof(Real),hipMemcpyDeviceToHost);
                hipMemcpy(max_ftotal0,d_max_ftotal0,sizeof(Real),hipMemcpyDeviceToHost);
                hipMemcpy(max_umag0,d_max_umag0,sizeof(Real),hipMemcpyDeviceToHost);

                hipcub::DeviceReduce::Max(dev_max_storage,max_storage_bytes,dt1,d_dt10,num_part3);
                hipDeviceSynchronize();
                hipMemcpy(dt10,d_dt10,sizeof(Real),hipMemcpyDeviceToHost);
                hipcub::DeviceReduce::Max(dev_max_storage,max_storage_bytes,dt2,d_dt20,num_part3);
                hipDeviceSynchronize();
                hipMemcpy(dt20,d_dt20,sizeof(Real),hipMemcpyDeviceToHost);
                hipcub::DeviceReduce::Max(dev_max_storage,max_storage_bytes,dt3,d_dt30,num_part3);
                hipDeviceSynchronize();
                hipMemcpy(dt30,d_dt30,sizeof(Real),hipMemcpyDeviceToHost);
                hipcub::DeviceReduce::Max(dev_max_storage,max_storage_bytes,dt4,d_dt40,num_part3);
                hipDeviceSynchronize();
                hipMemcpy(dt40,d_dt40,sizeof(Real),hipMemcpyDeviceToHost);
                hipcub::DeviceReduce::Max(dev_max_storage,max_storage_bytes,dt5,d_dt50,num_part3);
                hipDeviceSynchronize();
                hipMemcpy(dt50,d_dt50,sizeof(Real),hipMemcpyDeviceToHost);

                Real dt_delta = 0.1;
                Real dt_vel = 1.0/dt10[0];
                Real dt_ft = 0.1;
                Real dt_vis = 1.0/dt20[0];
                Real timestep[4] = {dt_delta,dt_vel,dt_ft,dt_vis};
                dt = min(timestep[0],timestep[1]);
                dt = min(dt,timestep[2]);
                dt = min(dt,timestep[3]);

                int_t integer=((time>=plotcount[0]*time_output)&&(time<(plotcount[0]+1)*time_output));

                if(integer) printf("dt_delta = %2.2e\ndt_vel = %2.2e\ndt_ft = %2.2e\ndt_vis = %2.2e\n",timestep[0],timestep[1],timestep[2],timestep[3]);
                if(integer) printf("dt = %e[s]\n",dt);

                // Set up ALE domain (update cell indices, alpha values)
                setALEdomain(b, t, dev_P1);

                // Perform neighbor search and particle sorting (NNPS)
                NNPS(
                    g_idx_in, g_idx, p_idx_in, p_idx, g_str, g_end,
                    dev_P1, dev_SP1, dev_P2, dev_SP2,
                    dev_sort_storage, &sort_storage_bytes,
                    b, t, s
                );

                // Enforce velocity boundary conditions (no-slip, penetration, etc.)
                velocityBC(b, t, g_str, g_end, dev_SP1);

                // Compute advection term (predictor step)
                advectionForce(b, t, g_str, g_end, dev_SP1, dev_SP2, dev_SP3);

                // Apply velocity projection
                projection(b, t, dev_SP1, dev_SP2, dev_SP3);

                // Prepare (gradient correction, variable updates) before pressure solve
                preparation(b, t, g_str, g_end, dev_SP1, dev_SP2, dev_SP3);

                // Set initial pressure boundary conditions (only at first step)
                if (count == 0)
                    pressureBC(b, t, g_str, g_end, dev_SP1, dev_SP2, dev_SP3);

                // Solve Pressure Poisson Equation (PPE)
                PPE(b, t, g_str, g_end, dev_SP1, dev_SP2, dev_SP3);

                // Enforce pressure boundary conditions (Neumann, Dirichlet, etc.)
                pressureBC(b, t, g_str, g_end, dev_SP1, dev_SP2, dev_SP3);

                //  Calculate pressure force and update acceleration
                pressureForce(b, t, g_str, g_end, dev_SP1, dev_SP2, dev_SP3);

                //  Update all time-dependent variables and particle positions
                timeUpdateProjection(b, t, dev_SP1, dev_P1, dev_SP2, dev_P2, dev_SP3);

                //  Apply particle shifting technique (PST) for improved regularity
                shifting(b, t, g_str, g_end, dev_P1, dev_SP2, dev_SP3);

                // Save particle data for visualization or further analysis
                saveOutput(
                    plotcount,
                    file_P1, file_P2, file_P3,
                    dev_P1, dev_SP2, dev_SP3,
                    time0, P0, P1, P2, P3
                );

                if(integer){
                    kernel_copy_max<<<b,t>>>(dev_P1,dev_SP2,dev_SP3,max_rho,max_ft,max_umag);
                    hipDeviceSynchronize();
                
                // Find Max Velocity & Force using CUB - TID=0
                hipcub::DeviceReduce::Max(dev_max_storage,max_storage_bytes,max_umag,d_max_umag0,num_part3);
                hipcub::DeviceReduce::Max(dev_max_storage,max_storage_bytes,max_rho,d_max_rho0,num_part3);
                hipcub::DeviceReduce::Max(dev_max_storage,max_storage_bytes,max_ft,d_max_ftotal0,num_part3);
                hipDeviceSynchronize();
                hipMemcpy(max_umag0,d_max_umag0,sizeof(Real),hipMemcpyDeviceToHost);
                hipMemcpy(max_rho0,d_max_rho0,sizeof(Real),hipMemcpyDeviceToHost);
                hipMemcpy(max_ftotal0,d_max_ftotal0,sizeof(Real),hipMemcpyDeviceToHost);
                printf("dt = %e[s]\n",dt);
                printf("%d\t compress_max=%5.2f%\tu_max=%5.2f\tftotal_max=%5.2f\n\n",count,max_rho0[0],max_umag0[0],max_ftotal0[0]);
            }
            time+=dt;
            count++;
        }

        
        if(time>=time_end) break;

    }
	
    clock_t end=clock();
    clock_t elapsed = end-start;
    double calctime=(double)(elapsed/CLOCKS_PER_SEC);
    printf("total calculation time = %f\n",calctime);

	//-------------------------------------------------------------------------------------------------
	// ##. Save Restart File
	//-------------------------------------------------------------------------------------------------

    save_restart(file_P1,file_P2,file_P3);
    hipMemcpy(file_P1,dev_SP1,num_part3*sizeof(part1),hipMemcpyDeviceToHost);
    hipMemcpy(file_P2,dev_SP2,num_part3*sizeof(part2),hipMemcpyDeviceToHost);
    hipMemcpy(file_P3,dev_SP3,num_part3*sizeof(part3),hipMemcpyDeviceToHost);
    free(file_P2);
    free(file_P3);

	//-------------------------------------------------------------------------------------------------
	// ##. Memory Free
	//-------------------------------------------------------------------------------------------------
	free(file_P1);
    free(max_umag0); free(max_rho0); free(max_ftotal0); free(max_phi0);
    free(dt10); free(dt20); free(dt30); free(dt40); free(dt50);
    free(HP1);

    hipFree(g_idx); hipFree(p_idx); hipFree(g_idx_in); hipFree(p_idx_in); hipFree(g_str); hipFree(g_end);
    hipFree(dev_P1); hipFree(dev_SP1); hipFree(dev_P2); hipFree(dev_SP2); hipFree(dev_SP3);
    hipFree(max_rho); hipFree(max_umag); hipFree(max_ft); hipFree(max_phi);
    hipFree(d_max_umag0); hipFree(d_max_rho0); hipFree(d_max_ftotal0); hipFree(d_max_phi0);
    hipFree(dt1); hipFree(dt2); hipFree(dt3); hipFree(dt4); hipFree(dt5);
    hipFree(d_dt10); hipFree(d_dt20); hipFree(d_dt30); hipFree(d_dt40); hipFree(d_dt50);
    hipFree(dev_sort_storage); hipFree(dev_max_storage);

    free(Cdp); free(Cdv); free(Cd);
    free(Clp); free(Clv); free(Cl);
    free(P0); free(P1); free(P2); free(P3); free(time0);
    free(plotcount);

    return 0;
}
